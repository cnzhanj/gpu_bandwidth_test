/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>

using namespace std;

const char *sSampleName = "P2P (Peer-to-Peer) GPU Bandwidth Latency Test";

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
        hipError_t e=hipGetLastError();                                 \
        if(e!=hipSuccess) {                                              \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }

void checkP2Paccess(int numGPUs)
{
    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);	//Set device to be used for GPU executions.

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            if (i!=j)
            {
                hipDeviceCanAccessPeer(&access,i,j);//	Queries if a device may directly access a peer device's memory.
                printf("Device=%d %s Access Peer Device=%d\n", i, access ? "CAN" : "CANNOT", j);
            }
        }
    }
    printf("\n***NOTE: In case a device doesn't have P2P access to other one, it falls back to normal memcopy procedure.\nSo you can see lesser Bandwidth (GB/s) in those cases.\n\n");
	//如果两块GPU卡之间可以直接访问内存，则显示CAN，否则，则显示CANNOT，并且最终的带宽矩阵差异将会比较大。
}

void enableP2P(int numGPUs)
{
    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            hipDeviceCanAccessPeer(&access,i,j);

            if (access)
            {
                hipDeviceEnablePeerAccess(j,0);
				/*
				peerDevice
					- Peer device to enable direct access to from the current device
				flags
					- Reserved for future use and must be set to 0
				*/
				cudaCheckError();
            }
        }
    }
}
void disableP2P(int numGPUs)
{
    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {
            int access;
            hipDeviceCanAccessPeer(&access, i, j);

            if (access)
            {
                hipDeviceDisablePeerAccess(j);
				/*
				Disables direct access to memory allocations on a peer device.
				peerDevice
					- Peer device to disable direct access to
				*/
                hipGetLastError();
            }
        }
    }
}

void outputBandwidthMatrix(int numGPUs)
{
    int numElems=10000000;//元素个数
    int repeat=5;			//重复5次
    vector<int *> buffers(numGPUs);//定义numGPUs个int*数组，也就是二维数组
    vector<hipEvent_t> start(numGPUs);//统计两次事件之间的时间
    vector<hipEvent_t> stop(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],numElems*sizeof(int));
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
    }

    vector<double> bandwidthMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {

            hipDeviceSynchronize();
            cudaCheckError();
            hipEventRecord(start[i]);

            for (int r=0; r<repeat; r++)
            {
                hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,sizeof(int)*numElems);
            }

            hipEventRecord(stop[i]);
            hipDeviceSynchronize();
            cudaCheckError();

            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);
            double time_s=time_ms/1e3;

            double gb=numElems*sizeof(int)*repeat/(double)1e9;
            bandwidthMatrix[i*numGPUs+j]=gb/time_s;
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
            printf("%6.02f ", bandwidthMatrix[i*numGPUs+j]);
        }

        printf("\n");
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
    }
}

void outputBidirectionalBandwidthMatrix(int numGPUs)
{
    int numElems=10000000;		//元素个数
    int repeat=5;				//重复5次
    vector<int *> buffers(numGPUs);		//定义numGPUs个int*数组，也就是二维数组
    vector<hipEvent_t> start(numGPUs);//统计两次事件之间的时间
    vector<hipEvent_t> stop(numGPUs);
    vector<hipStream_t> stream0(numGPUs);//定义流
    vector<hipStream_t> stream1(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],numElems*sizeof(int));	//给每个gpu分配显存，第一个参数之所以取地址，也就是二维指针，是因为buffer的地址是在
														//主存上，第一个参数传递的是cpu主存中指针变量的地址，函数完成后会向这个地址写入在GPU中分配的显存地址值
        cudaCheckError();
        hipEventCreate(&start[d]);		//创建事件
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
        hipStreamCreate(&stream0[d]);
        cudaCheckError();
        hipStreamCreate(&stream1[d]);
        cudaCheckError();
    }

    vector<double> bandwidthMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i); 

        for (int j=0; j<numGPUs; j++)
        {

            hipDeviceSynchronize();
            cudaCheckError();
            hipEventRecord(start[i]);

            for (int r=0; r<repeat; r++)
            {
                hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,sizeof(int)*numElems,stream0[i]);
                hipMemcpyPeerAsync(buffers[j],j,buffers[i],i,sizeof(int)*numElems,stream1[i]);
            }

            hipEventRecord(stop[i]);
            hipDeviceSynchronize();
            cudaCheckError();

            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);
            double time_s=time_ms/1e3;

            double gb=2.0*numElems*sizeof(int)*repeat/(double)1e9;
            bandwidthMatrix[i*numGPUs+j]=gb/time_s;
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
            printf("%6.02f ", bandwidthMatrix[i*numGPUs+j]);
        }

        printf("\n");
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
        hipStreamDestroy(stream0[d]);
        cudaCheckError();
        hipStreamDestroy(stream1[d]);
        cudaCheckError();
    }
}

void outputLatencyMatrix(int numGPUs)
{
    int repeat=10000;
    vector<int *> buffers(numGPUs);
    vector<hipEvent_t> start(numGPUs);
    vector<hipEvent_t> stop(numGPUs);

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipMalloc(&buffers[d],1);
        cudaCheckError();
        hipEventCreate(&start[d]);
        cudaCheckError();
        hipEventCreate(&stop[d]);
        cudaCheckError();
    }

    vector<double> latencyMatrix(numGPUs*numGPUs);

    for (int i=0; i<numGPUs; i++)
    {
        hipSetDevice(i);

        for (int j=0; j<numGPUs; j++)
        {

            hipDeviceSynchronize();
            cudaCheckError();
            hipEventRecord(start[i]);

            for (int r=0; r<repeat; r++)
            {
                hipMemcpyPeerAsync(buffers[i],i,buffers[j],j,1);
            }

            hipEventRecord(stop[i]);
            hipDeviceSynchronize();
            cudaCheckError();

            float time_ms;
            hipEventElapsedTime(&time_ms,start[i],stop[i]);

            latencyMatrix[i*numGPUs+j]=time_ms*1e3/repeat;
        }
    }

    printf("   D\\D");

    for (int j=0; j<numGPUs; j++)
    {
        printf("%6d ", j);
    }

    printf("\n");

    for (int i=0; i<numGPUs; i++)
    {
        printf("%6d ",i);

        for (int j=0; j<numGPUs; j++)
        {
            printf("%6.02f ", latencyMatrix[i*numGPUs+j]);
        }

        printf("\n");
    }

    for (int d=0; d<numGPUs; d++)
    {
        hipSetDevice(d);
        hipFree(buffers[d]);
        cudaCheckError();
        hipEventDestroy(start[d]);
        cudaCheckError();
        hipEventDestroy(stop[d]);
        cudaCheckError();
    }
}

int main(int argc, char **argv)
{

    int numGPUs;
    hipGetDeviceCount(&numGPUs);

    printf("[%s]\n", sSampleName);

    //output devices
    for (int i=0; i<numGPUs; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,i);
        printf("Device: %d, %s, pciBusID: %x, pciDeviceID: %x, pciDomainID:%x\n",i,prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID);
    }

    checkP2Paccess(numGPUs);

    //compute cliques
    vector<vector<int> > cliques;

    vector<bool> added(numGPUs,false);

    for (int i=0; i<numGPUs; i++)
    {
        if (added[i]==true)
            continue;         //already processed

        //create new clique with i
        vector<int> clique;
        added[i]=true;
        clique.push_back(i);

        for (int j=i+1; j<numGPUs; j++)
        {
            int access;
            hipDeviceCanAccessPeer(&access,i,j);

            if (access)
            {
                clique.push_back(j);
                added[j]=true;
            }
        }

        cliques.push_back(clique);
    }

    printf("P2P Cliques: \n");

    for (int c=0; c<(int)cliques.size(); c++)
    {
        printf("[");

        for (int j=0; j<(int)cliques[c].size()-1; j++)
        {
            printf("%d ",cliques[c][j]);
        }

        printf("%d]\n",cliques[c][cliques[c].size()-1]);
    }

    printf("Unidirectional P2P=Disabled Bandwidth Matrix (GB/s)\n");
    outputBandwidthMatrix(numGPUs);
    enableP2P(numGPUs);
    printf("Unidirectional P2P=Enabled Bandwidth Matrix (GB/s)\n");
    outputBandwidthMatrix(numGPUs);
    disableP2P(numGPUs);
    printf("Bidirectional P2P=Disabled Bandwidth Matrix (GB/s)\n");
    outputBidirectionalBandwidthMatrix(numGPUs);
    enableP2P(numGPUs);
    printf("Bidirectional P2P=Enabled Bandwidth Matrix (GB/s)\n");
    outputBidirectionalBandwidthMatrix(numGPUs);


    disableP2P(numGPUs);
    printf("P2P=Disabled Latency Matrix (us)\n");
    outputLatencyMatrix(numGPUs);
    enableP2P(numGPUs);
    printf("P2P=Enabled Latency Matrix (us)\n");
    outputLatencyMatrix(numGPUs);

    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    exit(EXIT_SUCCESS);
}
